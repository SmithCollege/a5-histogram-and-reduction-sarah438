#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 10

// used resource: chrome-extension://efaidnbmnnnibpcajpcglclefindmkaj/https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf

double get_clock() {
        struct timeval tv;
        int ok;
        ok = gettimeofday(&tv, (void *) 0);
        if (ok<0) {
                printf("gettimeofday error");
        }
        return (tv.tv_sec * 1.0 + tv.tv_usec * 1.0E-6);
}


__global__ void reducMax(int*input, int*output){
        __shared__ int partialMax[2*BLOCK_SIZE];
        unsigned int t = threadIdx.x;
        unsigned int i = blockIdx.x*blockDim.x+threadIdx.x;
        partialMax[t] = input[i];
        __syncthreads();
        for(unsigned int stride = 1; stride <=blockDim.x; stride *=2){
                        __syncthreads();
                        int indexM = 2*stride*t;
                if(indexM<blockDim.x){
                        if(partialMax[indexM] < partialMax[indexM+stride]){
                                partialMax[indexM] = partialMax[indexM+stride];
                        }
                }
        }
        if(t==0){
                output[blockIdx.x] = partialMax[0];
        }
}

__global__ void reducMin(int*input, int*output){
        __shared__ int partialMin[2*BLOCK_SIZE];
        unsigned int t = threadIdx.x;
        unsigned int i = blockIdx.x*blockDim.x+threadIdx.x;
        partialMin[t] = input[i];
        __syncthreads();
        for(unsigned int stride = 1; stride<=blockDim.x; stride*=2){
                __syncthreads();
                int indexm = 2*stride*t;
                if(indexm<blockDim.x){
                        if (partialMin[indexm] < partialMin[indexm+stride]){
                                partialMin[indexm] = partialMin[indexm+stride];
                        }
                }
        }
        if(t==0){
                output[blockIdx.x] = partialMin[0];
        }
}


__global__ void reducMult(int*input, int * output){
        __shared__ int partialProd[2*BLOCK_SIZE];
        unsigned int t = threadIdx.x;
        unsigned int i = blockIdx.x*blockDim.x+threadIdx.x;
        partialProd[t] = input[i];
        __syncthreads();
        for(unsigned int stride = 1; stride<=blockDim.x; stride*=2){
                __syncthreads();
                int indexP = 2*stride*t;
                if(indexP<blockDim.x){
                        partialProd[indexP] += partialProd[indexP+stride];
                }
        }
        if(t==0){
                output[blockIdx.x] = partialProd[0];
        }

}

__global__ void reducSum(int * input, int * output){
        __shared__ int partialSum[2*BLOCK_SIZE];

        //printf("%d \n", 3);
        //each thrad loads one element from global to shared memory
        unsigned int t = threadIdx.x;
        unsigned int i = blockIdx.x*blockDim.x+threadIdx.x;
        partialSum[t] = input[i];
        __syncthreads();
        //do reduction in shared memory
        for(unsigned int stride=1; stride<=blockDim.x; stride*=2){
                //if(tid%(2*s)==0){
                __syncthreads();
                int indexS = 2*stride*t;
                if(indexS<blockDim.x){
                        partialSum[indexS] += partialSum[indexS+stride];
                        //printf("%d \n", 7);
                }
        }
        //write result for this block to global memory
        if(t==0){
                output[blockIdx.x] = partialSum[0];
        }

}

__global__ void histo_kernal(unsigned int *input, int size, unsigned int *histo){
        int i = threadIdx.x + blockIdx.x*blockDim.x;

        // stride is the total number of threads
        int stride = blockDim.x * gridDim.x;

        // All threads in the grid collectively handle blockDim.x*gridDim.x consecutive elements
        while (i<size){
                atomicAdd(&(histo[input[i]]),1);
                i+=stride;
        }
}

int main() {
        double t0 = get_clock();

        int * input;
        int * output;
        int * histo;
        // allocate memory
        hipMallocManaged(&input,sizeof(int)*BLOCK_SIZE);
        hipMallocManaged(&output, sizeof(int)*BLOCK_SIZE);
        hipMallocManaged(&output, sizeof(int)*BLOCK_SIZE);
        //int* input = malloc(sizeof(int) * SIZE);
        //int* output = malloc(sizeof(int) * SIZE);

        int length = 0;
        // initialize inputs
        //srand(123);
        for (int i = 0; i < BLOCK_SIZE; i++) {
                //input[i] = rand() % 10;
                input[i] = i;
                length++;
        };

                reducSum<<<1,BLOCK_SIZE>>>(input, output);
                printf("%d \n", output[0]);
        // check results
        for (int i = 0; i < BLOCK_SIZE; i++) {
        printf("%d ", input[i]);
        }
        printf("\n");

        // free mem
        hipFree(input);
        hipFree(output);
        hipFree(histo);
        //free(output);

        double t1 = get_clock();
        printf("time per call: %f s\n", ((t1-t0)) );

        return 0;
}
