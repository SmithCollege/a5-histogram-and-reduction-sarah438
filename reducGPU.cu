
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define BLOCK_SIZE 10

// used resource: chrome-extension://efaidnbmnnnibpcajpcglclefindmkaj/https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf

double get_clock() {
        struct timeval tv;
        int ok;
        ok = gettimeofday(&tv, (void *) 0);
        if (ok<0) {
                printf("gettimeofday error");
        }
        return (tv.tv_sec * 1.0 + tv.tv_usec * 1.0E-6);
}


__global__ void reducMax(int*input, int*output){
        __shared__ int partialMax[2*BLOCK_SIZE];
        unsigned int t = threadIdx.x;
        unsigned int i = blockIdx.x*blockDim.x+threadIdx.x;
        partialMax[t] = input[i];
        __syncthreads();
        for(unsigned int stride = 1; stride <=blockDim.x; stride *=2){
                if(t%stride==0){
                        if(partialMax[2*t] >= partialMax[2*t+stride]){
                                partialMax[2*t] = partialMax[2*t];
                        }
                        else{
                                partialMax[2*t] = partialMax[2*t+stride];
                        }
                }
        }
        if(t==0){
                output[blockIdx.x] = partialMax[0];
        }
}

__global__ void reducMin(int*input, int*output){
        __shared__ int partialMin[2*BLOCK_SIZE];
        unsigned int t = threadIdx.x;
        unsigned int i = blockIdx.x*blockDim.x+threadIdx.x;
        partialMin[t] = input[i];
        __syncthreads();
        for(unsigned int stride = 1; stride<=blockDim.x; stride*=2){
                __syncthreads();
                if(t%stride==0){
                        if (partialMin[2*t] >= partialMin[2*t+stride]){
                                partialMin[2*t] = partialMin[2*t];
                        }
                        else{
                                partialMin[2*t] = partialMin[2*t+stride];
                        }

                }
        }
        if(t==0){
                output[blockIdx.x] = partialMin[0];
        }
}


__global__ void reducMult(int*input, int * output){
        __shared__ int partialProd[2*BLOCK_SIZE];
        unsigned int t = threadIdx.x;
        unsigned int i = blockIdx.x*blockDim.x+threadIdx.x;
        partialProd[t] = input[i];
        __syncthreads();
        for(unsigned int stride = 1; stride<=blockDim.x; stride*=2){
                __syncthreads();
                if(t%stride==0){
                        partialProd[2*t] += partialProd[2*t+stride];
                }
        }
        if(t==0){
                output[blockIdx.x] = partialProd[0];
        }

}

__global__ void reducSum(int * input, int * output){
        __shared__ int partialSum[2*BLOCK_SIZE];

        //printf("%d \n", 3);
        //each thrad loads one element from global to shared memory
        unsigned int t = threadIdx.x;
        unsigned int i = blockIdx.x*blockDim.x+threadIdx.x;
        partialSum[t] = input[i];
        __syncthreads();
        //do reduction in shared memory
        for(unsigned int stride=1; stride<=blockDim.x; stride*=2){
                //if(tid%(2*s)==0){
                __syncthreads();
                if(t %stride ==0){
                        partialSum[2*t] += partialSum[2*t+stride];
                        //printf("%d \n", 7);
                }
        }
        //write result for this block to global memory
        if(t==0){
                output[blockIdx.x] = partialSum[0];
        }

}

__global__ void histo_kernal(unsigned char *buffer, long size, unsigned int *histo){
        int i = threadIdx.x + blockIdx.x*blockDim.x;

        // stride is the total number of threads
        int stride = blockDim.x * gridDim.x;

        // All threads in the grid collectively handle blockDim.x*gridDim.x consecutive elements
        while (i<size){
                atomicAdd(&(histo[buffer[i]]),1);
                i+=stride;
        }
}

int main() {
        double t0 = get_clock();

                int * input;
                int * output;
        // allocate memory
        hipMallocManaged(&input,sizeof(int)*BLOCK_SIZE);
        hipMallocManaged(&output, sizeof(int)*BLOCK_SIZE);
        //int* input = malloc(sizeof(int) * SIZE);
        //int* output = malloc(sizeof(int) * SIZE);

        int length = 0;
        // initialize inputs
        //srand(123);
        for (int i = 0; i < BLOCK_SIZE; i++) {
                //input[i] = rand() % 10;
                input[i] = i;
                length++;
        };

                reducSum<<<1,BLOCK_SIZE>>>(input, output);
                printf("%d \n", output[0]);
        // check results
        for (int i = 0; i < BLOCK_SIZE; i++) {
        printf("%d ", input[i]);
        }
        printf("\n");

        // free mem
        hipFree(input);
        hipFree(output);
        //free(output);

        double t1 = get_clock();
        printf("time per call: %f s\n", ((t1-t0)) );

        return 0;
}
